#include "hip/hip_runtime.h"
#include "MeshKernel.h"

#include <sutil/vec_math.h>

#include <cuda/cuda_noise.cuh>

#include <hip/hip_math_constants.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.hpp>


#define NOISE_STRENGTH 0.5

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
	return (a + (b - 1)) / b;
}

__device__ float fBM(int numOctaves, float3 coordinate, float persistence, float scale, float low, float high, int seed)
{
	float maxAmp = 0;
	float amp = 1;
	float freq = scale;
	float noise = 0;

	// add successively smaller, higher - frequency terms
	for (int i = 0; i < numOctaves; ++i) {
		noise += cudaNoise::simplexNoise(coordinate, freq, seed) * amp;
		maxAmp += amp;
		amp *= persistence;
		freq *= 2;
	}

	// take the average value of the iterations
	noise /= maxAmp;

	// normalize the result
	noise = noise * (high - low) / 2 + (high + low) / 2;

	return noise;
}

__forceinline__ __device__ float3 calculateGerstnerWaveOffset(Wave* waves, int numWaves,
	float2 gridLocation, float t)
{
	float3 sum = make_float3(0.f);

	float L, wi, phi, rad, Qi, Ai, cosRad;
	float2 Di;
	for (int i = 0; i < numWaves; i++)
	{
		Qi = waves[i].steepness;
		Ai = waves[i].amplitude;
		L = waves[i].waveLength;
		wi = 2 / L;
		Di = make_float2(cos(waves[i].direction), sin(waves[i].direction));
		phi = waves[i].speed * 2 / L;
		rad = wi * dot(Di, gridLocation) + phi * t;
		cosRad = cos(rad);

		sum.x += Qi * Ai * Di.x * cosRad;
		sum.y += Ai * sin(rad);
		sum.z += Qi * Ai * Di.y * cosRad;
	}

	return sum;
}


__forceinline__ __device__ float3 calculateGerstnerWaveNormal(Wave* waves, int numWaves,
	float2 posPlane, float t)
{
	float3 sum = make_float3(0.f, 1.f, 0.f);

	float L, wi, phi, rad, Qi, Ai, WA, cosRad, sinRad;
	float2 Di;
	for (int i = 0; i < numWaves; i++)
	{
		Qi = waves[i].steepness;
		Ai = waves[i].amplitude;
		L = waves[i].waveLength;
		wi = 2 / L;
		WA = wi * Ai;
		Di = make_float2(cos(waves[i].direction), sin(waves[i].direction));
		phi = waves[i].speed * 2 / L;
		rad = wi * dot(Di, posPlane) + phi * t;
		cosRad = cos(rad);
		sinRad = sin(rad);

		sum.x += -Di.x * WA * cosRad;
		sum.y += -Qi * WA * sinRad;
		sum.z += -Di.y * WA * cosRad;
	}

	return normalize(sum);
}

__global__ void generateGridMesh(MeshBuffer meshBuffer, Wave* waves,
	int numWaves, int numSamplesX, int numSamplesZ, float length, float t)
{
	unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int ty = blockIdx.y * blockDim.y + threadIdx.y;

	int X = numSamplesX - 1;
	int Z = numSamplesZ - 1;
	if (tx > X || ty > Z) return;
	unsigned int indexVertex = tx * numSamplesZ + ty;

	float x0 = (tx - X / 2.0f) * length / X;
	float z0 = (ty - Z / 2.0f) * length / Z;
	float2 gridLocation = make_float2(x0, z0);

	float3 newPos = make_float3(gridLocation.x, 0.f, gridLocation.y) +
		calculateGerstnerWaveOffset(waves, numWaves, gridLocation, t);

	float noise = fBM(1, newPos, 0.5, 0.03, -NOISE_STRENGTH, NOISE_STRENGTH, 9);
	newPos.y += noise;

	meshBuffer.pos[indexVertex] = newPos;

	//meshBuffer.normal[indexVertex] = calculateGerstnerWaveNormal(waves,
	//	numWaves, make_float2(newPos.x, newPos.z), t);

	if (tx < X && ty < Z) {
		int indexIndices = 6 * (tx * X + ty);
		meshBuffer.indices[indexIndices] = indexVertex;
		meshBuffer.indices[indexIndices + 1] = indexVertex + numSamplesZ;
		meshBuffer.indices[indexIndices + 2] = indexVertex + numSamplesZ + 1;
		meshBuffer.indices[indexIndices + 3] = indexVertex;
		meshBuffer.indices[indexIndices + 4] = indexVertex + numSamplesZ + 1;
		meshBuffer.indices[indexIndices + 5] = indexVertex + 1;
	}
}

__global__ void calculateNormalDuDv(MeshBuffer meshBuffer, int numSamplesX, int numSamplesZ, float length)
{
	unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int ty = blockIdx.y * blockDim.y + threadIdx.y;

	int X = numSamplesX - 1;
	int Z = numSamplesZ - 1;
	if (tx > X || ty > Z) return;
	unsigned int indexVertex = tx * numSamplesZ + ty;

	float2 slope;
	float2 diff = make_float2(length / X * 2, length / Z * 2);
	if (tx > 0 && ty > 0 && tx < X && ty < Z)
	{
		float3 xp1 = meshBuffer.pos[(tx + 1) * numSamplesZ + ty];
		float3 xm1 = meshBuffer.pos[(tx - 1) * numSamplesZ + ty];
		float3 yp1 = meshBuffer.pos[tx * numSamplesZ + ty + 1];
		float3 ym1 = meshBuffer.pos[tx * numSamplesZ + ty - 1];

		slope.x = xp1.y - xm1.y;
		slope.y = yp1.y - ym1.y;

		diff.x = xp1.x - xm1.x;
		diff.y = yp1.z - ym1.z;
	}
	else {
		slope = make_float2(0.0f, 0.0f);
	}
	float3 normal = normalize(cross(make_float3(0.0f, slope.y, diff.y),
		make_float3(diff.x, slope.x, 0.0f)));

	meshBuffer.normal[indexVertex] = normal;
}

void cudaGenerateGridMesh(MeshBuffer meshBuffer, Wave* waves, int numWaves,
	int numSamplesX, int numSamplesZ, float length, float t)
{
	dim3 block(16, 16, 1);
	dim3 grid(cuda_iDivUp(numSamplesX, block.x), cuda_iDivUp(numSamplesZ, block.y), 1);

	generateGridMesh << <grid, block, 0, 0 >> > (meshBuffer, waves, numWaves,
		numSamplesX, numSamplesZ, length, t);

	calculateNormalDuDv << <grid, block, 0, 0 >> > (meshBuffer, numSamplesX, numSamplesZ, length);
}

__global__ void updateGridMesh(MeshBuffer meshBuffer, Wave* waves, int numWaves,
	int numSamplesX, int numSamplesZ, float length, float t)
{
	unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int ty = blockIdx.y * blockDim.y + threadIdx.y;

	int X = numSamplesX - 1;
	int Z = numSamplesZ - 1;
	if (tx > X || ty > Z) return;
	unsigned int indexVertex = tx * numSamplesZ + ty;

	float x0 = (tx - X / 2.0f) * length / X;
	float z0 = (ty - Z / 2.0f) * length / Z;
	float2 gridLocation = make_float2(x0, z0);

	float3 newPos = make_float3(gridLocation.x, 0.f, gridLocation.y) +
		calculateGerstnerWaveOffset(waves, numWaves, gridLocation, t);

	float noise = fBM(2, make_float3(gridLocation, t), 0.5, 0.15, -NOISE_STRENGTH, NOISE_STRENGTH, 9);
	newPos.y += noise;

	meshBuffer.pos[indexVertex] = newPos;

//	meshBuffer.normal[indexVertex] = calculateGerstnerWaveNormal(waves,
//		numWaves, make_float2(newPos.x, newPos.z), t);
}

void cudaUpdateGridMesh(MeshBuffer meshBuffer, Wave* waves, int numWaves,
	int numSamplesX, int numSamplesZ, float length, float t)
{
	dim3 block(16, 16, 1);
	dim3 grid(cuda_iDivUp(numSamplesX, block.x), cuda_iDivUp(numSamplesZ, block.y), 1);

	updateGridMesh << <grid, block, 0, 0 >> > (meshBuffer, waves, numWaves,
		numSamplesX, numSamplesZ, length, t);

	calculateNormalDuDv << <grid, block, 0, 0 >> > (meshBuffer, numSamplesX, numSamplesZ, length);
}
